#include "data_io.h"
#include "parameters.h"
#include "cuda_helpers.h"
#include "timing.h"
#include "bucket.h"
#include "grid_operations.h"
#include "constants.h"


int main(int argc, char** argv) {
    // Set CUDA device
    CUDA_CHECK(hipSetDevice(0));

    // Read data
    IO::ReadData();

    // Allocate and set simulation parameters
    mps::SimulationParameters params;
    params.SetParameters();
    params.AllocateBucket();

    // CUDA memory allocations
    double *d_acc;
    double *d_pos;
    double *d_vel;
    double *d_prs;
    double *d_pav;
    int *d_typ;
    int *d_bfst;
    int *d_blst;
    int *d_nxt;

    CUDA_CHECK(hipMalloc((void**)&d_typ, sizeof(int) * IO::nP));
    CUDA_CHECK(hipMalloc((void**)&d_acc, sizeof(double) * IO::nP * NUM_DIMENSIONS));
    CUDA_CHECK(hipMalloc((void**)&d_pos, sizeof(double) * IO::nP * NUM_DIMENSIONS));
    CUDA_CHECK(hipMalloc((void**)&d_vel, sizeof(double) * IO::nP * NUM_DIMENSIONS));
    CUDA_CHECK(hipMalloc((void**)&d_prs, sizeof(double) * IO::nP));
    CUDA_CHECK(hipMalloc((void**)&d_pav, sizeof(double) * IO::nP));
    CUDA_CHECK(hipMalloc((void**)&d_bfst, sizeof(int) * params.n_bxyz));
    CUDA_CHECK(hipMalloc((void**)&d_blst, sizeof(int) * params.n_bxyz));
    CUDA_CHECK(hipMalloc((void**)&d_nxt, sizeof(int) * IO::nP));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_typ, IO::typ, sizeof(int) * IO::nP, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_acc, IO::acc, sizeof(double) * IO::nP * NUM_DIMENSIONS, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_pos, IO::pos, sizeof(double) * IO::nP * NUM_DIMENSIONS, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_vel, IO::vel, sizeof(double) * IO::nP * NUM_DIMENSIONS, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_prs, IO::prs, sizeof(double) * IO::nP, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_pav, IO::pav, sizeof(double) * IO::nP, hipMemcpyHostToDevice));

    // Start timing
    double start_time;
    start_time = mps::GetTime();

    int total_threads;
    int blocks;

    while (TIM < params.kFinTim) {
        if (iLP % params.kOptFqc == 0) {
            CUDA_CHECK(hipMemcpy(IO::typ, d_typ, sizeof(int) * IO::nP, hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(IO::pos, d_pos, sizeof(double) * IO::nP * NUM_DIMENSIONS, hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(IO::vel, d_vel, sizeof(double) * IO::nP * NUM_DIMENSIONS, hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(IO::prs, d_prs, sizeof(double) * IO::nP, hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(IO::pav, d_pav, sizeof(double) * IO::nP, hipMemcpyDeviceToHost));

            CUDA_CHECK(hipMemset(d_pav, 0, sizeof(double) * IO::nP));

            mps::WriteData();

            int p_num = 0;
            for (int i = 0; i < IO::nP; i++) {
                if (typ[i] != mps::kGST) {
                    p_num++;
                }
            }
            printf("%5d th TIM: %lf / p_num: %d\n", iLP, TIM, p_num);

            if (TIM >= params.kFinTim) {
                break;
            }
        }

        total_threads = params.n_bxyz;
        blocks = total_threads / THREADS + 1;
        CUDA_CHECK(hipMemset(d_bfst, -1, sizeof(int) * params.n_bxyz));
        CUDA_CHECK(hipMemset(d_blst, -1, sizeof(int) * params.n_bxyz));
        CUDA_CHECK(hipMemset(d_nxt, -1, sizeof(int) * IO::nP));
        mps::MakeBucket<<<blocks, THREADS>>>(IO::nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos);
        CUDA_CHECK(hipDeviceSynchronize());

        total_threads = IO::nP;
        blocks = total_threads / THREADS + 1;

        mps::ComputeViscosityTerm<<<blocks, THREADS>>>(IO::nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos, d_vel, d_acc, params.r, params.a1);
        mps::UpdateParticles<<<blocks, THREADS>>>(IO::nP, d_typ, d_pos, d_vel, d_acc, d_prs);
        mps::CheckCollision<<<blocks, THREADS>>>(IO::nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos, d_vel, d_acc, params.dns, params.rlim2, params.col);
        CUDA_CHECK(hipMemcpy(d_vel, d_acc, sizeof(double) * nP * 3, hipMemcpyDeviceToDevice));

        mps::ComputePressure<<<blocks, THREADS>>>(IO::nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos, d_prs, params.dns, params.r, params.n0, params.a2);
        mps::ComputePressureGradient<<<blocks, THREADS>>>(IO::nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos, d_acc, d_prs, params.inv_dns, params.r, params.a3);
        mps::UpdateParticles2<<<blocks, THREADS>>>(nP, d_typ, d_pos, d_vel, d_acc, d_prs);
        mps::ComputePressure<<<blocks, THREADS>>>(nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos, d_prs, params.dns, params.r, params.n0, params.a2);
        mps::AddPressureArray<<<blocks, THREADS>>>(nP, d_pav, d_prs);

        iLP++;
        TIM += params.kDt;
    }

    // End timing
    double end_time;
    end_time = mps::GetTime();
    printf("Total: %13.6lf sec\n", end_time - start_time);

    // Free memory
    CUDA_CHECK(hipFree(d_typ));
    CUDA_CHECK(hipFree(d_acc));
    CUDA_CHECK(hipFree(d_pos));
    CUDA_CHECK(hipFree(d_vel));
    CUDA_CHECK(hipFree(d_prs));
    CUDA_CHECK(hipFree(d_pav));
    CUDA_CHECK(hipFree(d_bfst));
    CUDA_CHECK(hipFree(d_blst));
    CUDA_CHECK(hipFree(d_nxt));

    return 0;
}

