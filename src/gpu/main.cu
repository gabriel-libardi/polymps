#include "data_io.h"
#include "parameters.h"
#include "cuda_helpers.h"
#include "timing.h"
#include "bucket.h"
#include "grid_operations.h"


int main(int argc, char** argv) {
    // Set CUDA device
    CUDA_CHECK(hipSetDevice(0));

    // Read data
    mps::ReadData();

    // Allocate and set simulation parameters
    mps::SimulationParameters params;
    params.SetParameters();
    params.AllocateBucket();

    // CUDA memory allocations
    double *d_acc;
    double *d_pos;
    double *d_vel;
    double *d_prs;
    double *d_pav;
    int *d_typ;
    int *d_bfst;
    int *d_blst;
    int *d_nxt;

    CUDA_CHECK(hipMalloc((void**)&d_typ, sizeof(int) * nP));
    CUDA_CHECK(hipMalloc((void**)&d_acc, sizeof(double) * nP * 3));
    CUDA_CHECK(hipMalloc((void**)&d_pos, sizeof(double) * nP * 3));
    CUDA_CHECK(hipMalloc((void**)&d_vel, sizeof(double) * nP * 3));
    CUDA_CHECK(hipMalloc((void**)&d_prs, sizeof(double) * nP));
    CUDA_CHECK(hipMalloc((void**)&d_pav, sizeof(double) * nP));
    CUDA_CHECK(hipMalloc((void**)&d_bfst, sizeof(int) * params.n_bxyz));
    CUDA_CHECK(hipMalloc((void**)&d_blst, sizeof(int) * params.n_bxyz));
    CUDA_CHECK(hipMalloc((void**)&d_nxt, sizeof(int) * nP));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_typ, typ, sizeof(int) * nP, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_acc, acc, sizeof(double) * nP * 3, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_pos, pos, sizeof(double) * nP * 3, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_vel, vel, sizeof(double) * nP * 3, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_prs, prs, sizeof(double) * nP, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_pav, pav, sizeof(double) * nP, hipMemcpyHostToDevice));

    // Start timing
    double start_time;
    start_time = mps::GetTime();

    int total_threads;
    int blocks;

    while (TIM < params.kFinTim) {
        if (iLP % params.kOptFqc == 0) {
            CUDA_CHECK(hipMemcpy(typ, d_typ, sizeof(int) * nP, hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(pos, d_pos, sizeof(double) * nP * 3, hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(vel, d_vel, sizeof(double) * nP * 3, hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(prs, d_prs, sizeof(double) * nP, hipMemcpyDeviceToHost));
            CUDA_CHECK(hipMemcpy(pav, d_pav, sizeof(double) * nP, hipMemcpyDeviceToHost));

            CUDA_CHECK(hipMemset(d_pav, 0, sizeof(double) * nP));

            mps::WriteData();

            int p_num = 0;
            for (int i = 0; i < nP; i++) {
                if (typ[i] != mps::kGST) {
                    p_num++;
                }
            }
            printf("%5d th TIM: %lf / p_num: %d\n", iLP, TIM, p_num);

            if (TIM >= params.kFinTim) {
                break;
            }
        }

        total_threads = params.n_bxyz;
        blocks = total_threads / THREADS + 1;
        CUDA_CHECK(hipMemset(d_bfst, -1, sizeof(int) * params.n_bxyz));
        CUDA_CHECK(hipMemset(d_blst, -1, sizeof(int) * params.n_bxyz));
        CUDA_CHECK(hipMemset(d_nxt, -1, sizeof(int) * nP));
        mps::MakeBucket<<<blocks, THREADS>>>(nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos);
        CUDA_CHECK(hipDeviceSynchronize());

        total_threads = nP;
        blocks = total_threads / THREADS + 1;
        mps::ComputeViscosityTerm<<<blocks, THREADS>>>(nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos, d_vel, d_acc, params.r, params.a1);
        CUDA_CHECK(hipDeviceSynchronize());

        mps::UpdateParticles<<<blocks, THREADS>>>(nP, d_typ, d_pos, d_vel, d_acc, d_prs);
        CUDA_CHECK(hipDeviceSynchronize());

        mps::CheckCollision<<<blocks, THREADS>>>(nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos, d_vel, d_acc, params.dns, params.rlim2, params.col);
        CUDA_CHECK(hipMemcpy(d_vel, d_acc, sizeof(double) * nP * 3, hipMemcpyDeviceToDevice));
        CUDA_CHECK(hipDeviceSynchronize());

        mps::ComputePressure<<<blocks, THREADS>>>(nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos, d_prs, params.dns, params.r, params.n0, params.a2);
        CUDA_CHECK(hipDeviceSynchronize());

        mps::ComputePressureGradient<<<blocks, THREADS>>>(nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos, d_acc, d_prs, params.inv_dns, params.r, params.a3);
        CUDA_CHECK(hipDeviceSynchronize());

        mps::UpdateParticles2<<<blocks, THREADS>>>(nP, d_typ, d_pos, d_vel, d_acc, d_prs);
        CUDA_CHECK(hipDeviceSynchronize());

        mps::ComputePressure<<<blocks, THREADS>>>(nP, params.n_bx, params.n_bxy, params.n_bxyz, params.db_inv, d_bfst, d_blst, d_nxt, d_typ, d_pos, d_prs, params.dns, params.r, params.n0, params.a2);
        CUDA_CHECK(hipDeviceSynchronize());

        mps::AddPressureArray<<<blocks, THREADS>>>(nP, d_pav, d_prs);
        CUDA_CHECK(hipDeviceSynchronize());

        iLP++;
        TIM += params.kDt;
    }

    // End timing
    double end_time;
    end_time = mps::GetTime();
    printf("Total: %13.6lf sec\n", end_time - start_time);

    // Free memory
    CUDA_CHECK(hipFree(d_typ));
    CUDA_CHECK(hipFree(d_acc));
    CUDA_CHECK(hipFree(d_pos));
    CUDA_CHECK(hipFree(d_vel));
    CUDA_CHECK(hipFree(d_prs));
    CUDA_CHECK(hipFree(d_pav));
    CUDA_CHECK(hipFree(d_bfst));
    CUDA_CHECK(hipFree(d_blst));
    CUDA_CHECK(hipFree(d_nxt));

    return 0;
}

