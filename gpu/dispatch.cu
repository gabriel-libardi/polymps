#include <iostream>
#include <hip/hip_runtime.h>

#define N 3  // Size of the square matrices

// CUDA kernel to multiply two matrices
__global__ void matrixMul(float *a, float *b, float *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row < n && col < n) {
        for (int i = 0; i < n; ++i) {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main() {
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;
    int size = N * N * sizeof(float);

    // Allocate memory for host matrices
    a = (float *)malloc(size);
    b = (float *)malloc(size);
    c = (float *)malloc(size);

    // Initialize host matrices
    for (int i = 0; i < N * N; ++i) {
        a[i] = i + 1;
        b[i] = i + 1;
    }

    // Allocate memory for device matrices
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy host matrices to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 dimGrid(1, 1);
    dim3 dimBlock(N, N);

    // Launch the kernel
    matrixMul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print result matrix
    for (int i = 0; i < N * N; ++i) {
        std::cout << c[i] << " ";
        if ((i + 1) % N == 0) {
            std::cout << std::endl;
        }
    }

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}

